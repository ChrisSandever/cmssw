#include "hip/hip_runtime.h"
#include <cstdlib>
#include <limits>

#include <hip/hip_runtime.h>

#include "DataFormats/EcalDigi/interface/EcalDataFrame_Ph2.h"
#include "DataFormats/EcalDigi/interface/EcalLiteDTUSample.h"
#include "DataFormats/EcalRecHit/interface/EcalUncalibratedRecHit.h"
#include "FWCore/Utilities/interface/CMSUnrollLoop.h"
#include "DataFormats/EcalDigi/interface/EcalConstants.h"


#include "EcalUncalibRecHitPhase2WeightsKernels.h"
#include "KernelHelpers.h"

#include "EigenMatrixTypes_gpu.h"

#include "DeclsForKernelsPh2WeightsGPU.h"

// Kernel which executes weights algorithm on device

namespace ecal {
  namespace weights {

    __global__ void Phase2WeightsKernel(uint16_t const* digis_in,
                                        uint32_t const* dids,
                                        ::ecal::reco::StorageScalarType* amplitudeEB,
                                        uint32_t* dids_outEB,
                                        int const nchannels,
                                        double* weights_d,
                                        uint32_t* flagsEB
                                        // ,uint16_t* debug_d
                                        ) {

    constexpr int nsamples = EcalDataFrame_Ph2::MAXSAMPLES;
    int const tx = threadIdx.x + blockIdx.x * blockDim.x;

    if (tx < nchannels) {
      auto const did = DetId{dids[tx]};

      double amp = 0.0;
      bool g1 = false;

      float gains[2] = {10., 1.}; //since ecalPh2::gains doesn't work
      double gratio = 0.0;

      // CMS_UNROLL_LOOP
      for (int sample = 0; sample < nsamples; ++sample) {
        double adc = 1.0 * ecalLiteDTU::adc(digis_in[tx * nsamples + sample]);

        int gainId = ecalLiteDTU::gainId(digis_in[tx * nsamples + sample]);  // is the gain Id added properly to the digis in the first place?
        // gratio = ecalPh2::gains[gainId];   this gives error undefined in device code, hence it is hard coded above
        gratio = gains[gainId];
        if (gainId == 1) {
          g1= true;
        }
          // comment this out to time overhead
        amp = amp + (adc * gratio * weights_d[sample]); //weights_d might not have been copied properly?

        //  uncomment this and comment above to time overhead
    //       int gainId =0;
    //       amp = 0;
    //       double dummy = 0.;
    // for (size_t i = 0; i < 200000; ++i) {
    //   if (i % 2 == 0) {
    //     dummy += static_cast<double>(i);
    //   } else {
    //     dummy -= static_cast<double>(i);
    //   }
    // }
    // amp += dummy;



        }
      
      // debugging============================
      // if(tx < 16){
      //   debug_d[tx] = amp;
      // }
      // //

      amplitudeEB[tx] = amp;
      // chi2EB = 0.;
      // g_pedestalEB = 0.;
      dids_outEB[tx] = did.rawId();
      flagsEB = 0;
      if (g1) {
          flagsEB[tx] = EcalUncalibratedRecHit::kHasSwitchToGain1;
        }

      }

    }


  } //namespace weights
}  //namespace ecal