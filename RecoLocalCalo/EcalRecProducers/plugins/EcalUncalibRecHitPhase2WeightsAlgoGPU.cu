#include <iostream>
#include <limits>

#include <hip/hip_runtime.h>
#include "DataFormats/EcalDigi/interface/EcalDataFrame_Ph2.h"

#include "CondFormats/EcalObjects/interface/EcalMGPAGainRatio.h"
#include "DataFormats/EcalDigi/interface/EcalDataFrame.h"
#include "DataFormats/EcalDigi/interface/EcalDigiCollections.h"

#include "EcalUncalibRecHitPhase2WeightsKernels.h"
#include "EcalUncalibRecHitPhase2WeightsAlgoGPU.h"

#include "EigenMatrixTypes_gpu.h"

#include "DeclsForKernelsPh2WeightsGPU.h"

// entrypoint to kernal execution

//#define DEBUG

//#define ECAL_RECO_CUDA_DEBUG

namespace ecal {
  namespace weights {

    void entryPoint(ecal::DigisCollection<calo::common::DevStoragePolicy> const& ebDigis,
                    EventOutputDataGPUWeights& eventOutputGPU,
                    cms::cuda::device::unique_ptr<double[]>& weights_d,
                    hipStream_t cudaStream) {
      unsigned int totalChannels = ebDigis.size;

      unsigned int nchannels_per_block = 64;
      unsigned int threads_1d = nchannels_per_block;
      unsigned int blocks_1d = (totalChannels / threads_1d) + 1;

      int shared_bytes = 2 * sizeof(float) + EcalDataFrame_Ph2::MAXSAMPLES * sizeof(double) +
                         nchannels_per_block * (EcalDataFrame_Ph2::MAXSAMPLES * (sizeof(uint16_t)) + sizeof(float));

      Phase2WeightsKernel<<<blocks_1d, threads_1d, shared_bytes, cudaStream>>>(
          ebDigis.data.get(),
          ebDigis.ids.get(),
          eventOutputGPU.recHitsEB.amplitude.get(),
          eventOutputGPU.recHitsEB.amplitudeError.get(),
          eventOutputGPU.recHitsEB.did.get(),
          totalChannels,
          weights_d.get(),
          eventOutputGPU.recHitsEB.flags.get());
      cudaCheck(hipGetLastError());
    }

  }  // namespace weights
}  // namespace ecal
