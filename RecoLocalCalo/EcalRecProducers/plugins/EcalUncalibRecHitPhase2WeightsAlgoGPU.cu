#include <iostream>
#include <limits>

#include <hip/hip_runtime.h>
#include "DataFormats/EcalDigi/interface/EcalDataFrame_Ph2.h"

#include "CondFormats/EcalObjects/interface/EcalMGPAGainRatio.h"
#include "DataFormats/EcalDigi/interface/EcalDataFrame.h"
#include "DataFormats/EcalDigi/interface/EcalDigiCollections.h"

#include "EcalUncalibRecHitPhase2WeightsKernels.h"
#include "EcalUncalibRecHitPhase2WeightsAlgoGPU.h"

#include "EigenMatrixTypes_gpu.h"

#include "DeclsForKernelsPh2WeightsGPU.h"

// entrypoint to kernal execution

//#define DEBUG

//#define ECAL_RECO_CUDA_DEBUG

namespace ecal
{
  namespace weights
  {

    void entryPoint(ecal::DigisCollection<calo::common::DevStoragePolicy> const& ebDigis,
                    EventOutputDataGPUWeights& eventOutputGPU,
                    cms::cuda::device::unique_ptr<double[]>& weights_d,
                    // cms::cuda::device::unique_ptr<double[]>& debug_d,
                    hipStream_t cudaStream)
    {
      // using digis_type = std::vector<uint16_t>;
      // using dids_type = std::vector<uint32_t>;

      unsigned int totalChannels = ebDigis.size ;

    
      unsigned int nchannels_per_block = 32;
      unsigned int threads_1d = 10 * nchannels_per_block;
      unsigned int blocks_1d = (totalChannels / threads_1d) + 1;
      // unsigned int blocks_1d = 192;
      
      // unsigned int blocks_1d = threads_1d > 10 * totalChannels ? 1 : (totalChannels * 10 + threads_1d - 1) / threads_1d;
      int shared_bytes = nchannels_per_block * EcalDataFrame_Ph2::MAXSAMPLES * (sizeof(bool) + sizeof(bool) + sizeof(bool) + sizeof(bool) + sizeof(char) + sizeof(bool));

     
      Phase2WeightsKernel <<<blocks_1d, threads_1d, shared_bytes, cudaStream>>> (ebDigis.data.get(), 
                                                                  ebDigis.ids.get(),
                                                                  eventOutputGPU.recHitsEB.amplitude.get(),
                                                                  eventOutputGPU.recHitsEB.did.get(),
                                                                  totalChannels,
                                                                  weights_d.get(),
                                                                  eventOutputGPU.recHitsEB.flags.get()
                                                                  // ,debug_d.get()
                                                                  );
      cudaCheck(hipGetLastError());

    


    }

  } // namespace weights
}   // namespace ecal